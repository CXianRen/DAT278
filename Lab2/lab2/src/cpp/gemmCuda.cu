#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

//CUDA RunTime API
#include <hip/hip_runtime.h>

#include "gemmCuda.cuh"
#define TYPE float
#define BLOCK_SIZE 32


__global__ void gemmCUDA(TYPE* a, TYPE* b, TYPE* c, int n_row, int n_col, int n)
{
    /*Write your gemm kernel here*/
}


void gemm_cuda (TYPE * a_ptr, TYPE * b_ptr, TYPE * c_ptr,int a_row,int b_col, int n)
 {
    
    TYPE *cuda_a, *cuda_b, *cuda_c;

    auto a_mem_size = a_row*n;
    auto b_mem_size = n*b_col;
    auto c_mem_size = a_row*b_col;

    /* Task: Memory Allocation */
    hipMalloc(&cuda_a, sizeof(TYPE)* a_mem_size);
    hipMalloc(&cuda_b, sizeof(TYPE)* b_mem_size);
    hipMalloc(&cuda_c, sizeof(TYPE)* c_mem_size);

    /* Task: CUDA Memory Copy from Host to Device */
    hipMemcpy(cuda_a, a_ptr, sizeof(TYPE)*a_mem_size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b_ptr, sizeof(TYPE)*b_mem_size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_c, c_ptr, sizeof(TYPE)*c_mem_size, hipMemcpyHostToDevice);


    // Compute GEMM

    // change these and check if this works 
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid((b_col+ dimBlock.x - 1) / dimBlock.x, (a_row + dimBlock.y - 1) / dimBlock.y);
    

    gemmCUDA <<<dimGrid, dimBlock>>> (cuda_a, cuda_b,  cuda_c, a_row, b_col, n); 
    

    hipMemcpy(c_ptr, cuda_c, sizeof(TYPE)*c_mem_size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    

    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);
    


    
}

